#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <algorithm>
#include <omp.h>

const int nIters = 1000;

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

#define SYNC() \
  for(int d=0;d<numDev;d++) {                                         \
    hipSetDevice(d);                                                 \
    hipDeviceSynchronize();                                          \
    cudaCheckError();                                                 \
  }                                                                   \
  hipSetDevice(0);


void time_solve(int ROWS, int COLS, int numDev);

#define IDX(row,col,LDA) ( (row)*(LDA)+(col))
__global__ void simpleLaplaceIter_kernel(int ROWS, int COLS, double * Ain, double * Aout)
{
  int lda=COLS+2;
  int col=blockIdx.x*blockDim.x+threadIdx.x+1; //Pad to skip BC
  int row=blockIdx.y*blockDim.y+threadIdx.y+1; //Pad to skip BC

  if(row<=ROWS && col<=COLS) { //Ensure i and j are not BCs
    Aout[IDX(row,col,lda)]= .25 * 
      (  
       + Ain[IDX(row  ,col-1,lda)] 
       + Ain[IDX(row  ,col+1,lda)] 
       + Ain[IDX(row-1,col  ,lda)] 
       + Ain[IDX(row+1,col  ,lda)] 
      );
  }
}

int main(int argc, char** argv)
{
  printf("OMP Threads: %d\n", omp_get_max_threads()); 
  int numDev; hipGetDeviceCount(&numDev); printf("Num GPUs: %d\n", numDev);

  for(int i=0;i<numDev;i++) {
    hipSetDevice(i);
    for(int j=0;j<numDev;j++) {
      if(i!=j) {
        int access;
        hipDeviceCanAccessPeer(&access,i,j);
        if(access) {
          hipDeviceEnablePeerAccess(j,0);
          //printf("Peer Access %d -> %d\n",i,j);
          cudaCheckError();
        }
      }
    }
  }

  time_solve(2048, 2048, numDev);
  time_solve(4096, 4096, numDev);
  //time_solve(8192, 8192, numDev);

  printf("done\n");
  return 0;
}

double laplace_solve_reference(double *A, int ROWS, int COLS) {
  int LDA=COLS+2;
  size_t bytes=(ROWS+2)*(LDA)*sizeof(double);  //Pad for BCs
  int numDev=1;
  hipSetDevice(0);
  
  double *d_Ain;
  double *d_Aout;
  //allocate device memory
  hipMalloc(&d_Ain,bytes);
  hipMalloc(&d_Aout,bytes);
  cudaCheckError();

  //copy initial conditions to both buffers
  hipMemcpy(d_Ain,A,bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Aout,d_Ain,bytes,hipMemcpyDeviceToDevice);
  cudaCheckError();
  
  //start timer
  SYNC();
  double start=omp_get_wtime();
  
  dim3 blockSize(16,16);
  dim3 gridSize( ceil((double)(COLS)/blockSize.x), ceil((double)(ROWS)/blockSize.y));
  //perform iterative solve
  for(int i=0; i<nIters; i++)
  {
    simpleLaplaceIter_kernel<<<gridSize,blockSize>>>(ROWS,COLS,d_Ain,d_Aout);
    cudaCheckError();
    std::swap(d_Ain,d_Aout);
  }
 
  //stop timer
  SYNC();
  double end=omp_get_wtime();
  
  //copy results back to host
  hipMemcpy(A,d_Ain,bytes,hipMemcpyDeviceToHost);
  cudaCheckError();
  
  //free device memory
  hipFree(d_Ain);
  hipFree(d_Aout);
  cudaCheckError();

  return (end-start);
}

#include "laplace.h" 

void time_solve(int ROWS, int COLS, int numDev) {
  size_t bytes=(ROWS+2)*(COLS+2)*sizeof(double);  //Pad for BCs
  double *A, *Aserial;
  //Allocate pinned host memory
  hipHostMalloc(&A,bytes);
  hipHostMalloc(&Aserial,bytes);
  cudaCheckError();

  int LDA=COLS+2;
  //set initial conditions
  for(int i=0;i<ROWS+2;i++)
    for(int j=0;j<COLS+2;j++)
      Aserial[IDX(i,j,LDA)]=A[IDX(i,j,LDA)]=(i==0 || j==0 || i==ROWS+1 || j==COLS+1) ? 1 : 0;

  double time_s;
  double time_p;
  //solve in serial
  time_s=laplace_solve_reference(Aserial,ROWS,COLS);
  //solve in parallel
  time_p=laplace_solve(A,ROWS,COLS,numDev);

#if 0
  printf("Serial:\n");
  for(int i=0;i<ROWS+2;i++) {
    for(int j=0;j<COLS+2;j++) {
      printf("%6.2lg ",Aserial[IDX(i,j,LDA)]);
    }
    printf("\n");
  }
  printf("\n");
  printf("\n");
  printf("Parallel:\n");
  for(int i=0;i<ROWS+2;i++) {
    for(int j=0;j<COLS+2;j++) {
      printf("%6.2lg ",A[IDX(i,j,LDA)]);
    }
    printf("\n");
  }
  printf("\n");
  printf("\n");
#endif

  //verify answers are identical
  for(int i=0;i<ROWS+2;i++){
    for(int j=0;j<COLS+2;j++){
      if(Aserial[IDX(i,j,LDA)]!=A[IDX(i,j,LDA)]) {
        printf("%dx%d: Error solutions do not match at i: %d, j: %d\n", ROWS,COLS,i,j);
        goto cleanup;
      }
    }
  }
  

cleanup:
  printf("%dx%d: solve time, 1 GPU: %lg s, %d GPUs: %lg s, speedup: %lg, efficiency: %lg%%\n", ROWS, COLS, time_s, numDev, time_p, time_s/time_p, time_s/time_p/numDev*100);
  //free host memory
  hipHostFree(A);
  hipHostFree(Aserial);
  cudaCheckError();
}
